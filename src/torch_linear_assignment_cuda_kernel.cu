#include "hip/hip_runtime.h"
/*
  Implementation is based on the algorihtm presented in pages 1685-1686 of:

  DF Crouse. On implementing 2D rectangular assignment algorithms.
    IEEE Transactions on Aerospace and Electronic Systems
    52(4):1679-1696, August 2016
    doi: 10.1109/TAES.2016.140952
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include <torch/extension.h>

#include <limits>


typedef unsigned int uint32_t;


template <typename scalar_t>
__device__ __forceinline__
void array_fill(scalar_t *start, scalar_t *stop, scalar_t value) {
  for (; start < stop; ++start) {
    *start = value;
  }
}


template <typename scalar_t>
__device__ __forceinline__
uint32_t augmenting_path_cuda(uint32_t nr, uint32_t nc, uint32_t i,
			      scalar_t *cost, scalar_t *u, scalar_t *v,
			      uint32_t *path, uint32_t *row4col,
			      scalar_t *shortestPathCosts,
			      bool *SR, bool *SC,
			      uint32_t *remaining,
			      scalar_t *p_minVal,
			      scalar_t infinity)
{
    scalar_t minVal = 0;
    uint32_t num_remaining = nc;
    for (uint32_t it = 0; it < nc; it++) {
        remaining[it] = nc - it - 1;
    }

    array_fill(SR, SR + nr, false);
    array_fill(SC, SC + nc, false);
    array_fill(shortestPathCosts, shortestPathCosts + nc, infinity);

    uint32_t sink = -1;
    while (sink == -1) {
        uint32_t index = -1;
        scalar_t lowest = infinity;
        SR[i] = true;

        for (uint32_t it = 0; it < num_remaining; it++) {
            uint32_t j = remaining[it];
            scalar_t r = minVal + cost[i * nc + j] - u[i] - v[j];
            if (r < shortestPathCosts[j]) {
	      path[j] = i;
	      shortestPathCosts[j] = r;
            }
            if (shortestPathCosts[j] < lowest ||
                (shortestPathCosts[j] == lowest && row4col[j] == -1)) {
                lowest = shortestPathCosts[j];
                index = it;
            }
        }

        minVal = lowest;
        if (minVal == infinity) {
            return -1;
        }

        uint32_t j = remaining[index];
        if (row4col[j] == -1) {
            sink = j;
        } else {
            i = row4col[j];
        }

        SC[j] = true;
        remaining[index] = remaining[--num_remaining];
    }
    *p_minVal = minVal;
    return sink;
}


template <typename scalar_t, typename index_t>
__device__ __forceinline__
void solve_cuda_kernel(uint32_t nr, uint32_t nc,
		       scalar_t *cost, index_t *matching,
		       scalar_t *u, scalar_t *v,
		       scalar_t *shortestPathCosts,
		       uint32_t *path, uint32_t *col4row, uint32_t *row4col,
		       bool *SR, bool *SC,
		       uint32_t *remaining,
		       scalar_t infinity)
{
  array_fill(u, u + nr, (scalar_t) 0);
  array_fill(v, v + nc, (scalar_t) 0);
  array_fill(path, path + nc, (uint32_t) -1);
  array_fill(row4col, row4col + nc, (uint32_t) -1);
  array_fill(col4row, col4row + nr, (uint32_t) -1);

  scalar_t minVal;
  for (uint32_t curRow = 0; curRow < nr; ++curRow) {
    auto sink = augmenting_path_cuda(nr, nc, curRow, cost,
				     u, v,
				     path, row4col,
				     shortestPathCosts,
				     SR, SC,
				     remaining,
				     &minVal, infinity);

    CUDA_KERNEL_ASSERT(sink >= 0 && "Infeasible matrix");

    u[curRow] += minVal;
    for (uint32_t i = 0; i < nr; i++) {
      if (SR[i] && i != curRow) {
	u[i] += minVal - shortestPathCosts[col4row[i]];
      }
    }

    for (uint32_t j = 0; j < nc; j++) {
      if (SC[j]) {
	v[j] -= minVal - shortestPathCosts[j];
      }
    }

    uint32_t i;
    uint32_t j = sink;
    uint32_t swap;
    while (1) {
      i = path[j];
      row4col[j] = i;
      swap = j;
      j = col4row[i];
      col4row[i] = swap;
      if (i == curRow) {
	break;
      }
    }
  }

  for (uint32_t i = 0; i < nr; i++) {
    matching[i] = col4row[i];
  }
}


template <typename scalar_t, typename index_t>
__global__
void solve_cuda_kernel_batch(uint32_t bs, uint32_t nr, uint32_t nc,
			     scalar_t *cost, index_t *matching,
			     scalar_t *u, scalar_t *v,
			     scalar_t *shortestPathCosts,
			     uint32_t *path, uint32_t *col4row, uint32_t *row4col,
			     bool *SR, bool *SC,
			     uint32_t *remaining,
			     scalar_t infinity) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= bs) {
    return;
  }

  solve_cuda_kernel(nr, nc,
		    cost + i * nr * nc,
		    matching + i * nr,
		    u + i * nr,
		    v + i * nc,
		    shortestPathCosts + i * nc,
		    path + i * nc,
		    col4row + i * nr,
		    row4col + i * nc,
		    SR + i * nr,
		    SC + i * nc,
		    remaining + i * nc,
		    infinity);
}


template <typename scalar_t, typename index_t>
void solve_cuda_batch(uint32_t bs, uint32_t nr, uint32_t nc,
		      scalar_t *cost, index_t *matching) {
  TORCH_CHECK(std::numeric_limits<scalar_t>::has_infinity, "Data type doesn't have infinity.");
  auto infinity = std::numeric_limits<scalar_t>::infinity();

  thrust::device_vector<scalar_t> u(bs * nr);
  thrust::device_vector<scalar_t> v(bs * nc);
  thrust::device_vector<scalar_t> shortestPathCosts(bs * nc);
  thrust::device_vector<uint32_t> path(bs * nc);
  thrust::device_vector<uint32_t> col4row(bs * nr);
  thrust::device_vector<uint32_t> row4col(bs * nc);
  thrust::device_vector<bool> SR(bs * nr);
  thrust::device_vector<bool> SC(bs * nc);
  thrust::device_vector<uint32_t> remaining(bs * nc);

  int blockSize;
  int minGridSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
				     (void *) solve_cuda_kernel_batch<scalar_t, index_t>,
				     0, bs);

  int gridSize = (bs + blockSize - 1) / blockSize;
  solve_cuda_kernel_batch<<<gridSize, blockSize>>>(
    bs, nr, nc,
    cost, matching,
    thrust::raw_pointer_cast(&u.front()),
    thrust::raw_pointer_cast(&v.front()),
    thrust::raw_pointer_cast(&shortestPathCosts.front()),
    thrust::raw_pointer_cast(&path.front()),
    thrust::raw_pointer_cast(&col4row.front()),
    thrust::raw_pointer_cast(&row4col.front()),
    thrust::raw_pointer_cast(&SR.front()),
    thrust::raw_pointer_cast(&SC.front()),
    thrust::raw_pointer_cast(&remaining.front()),
    infinity);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    TORCH_CHECK(false, hipGetErrorString(err));
  }
}


torch::Tensor batch_linear_assignment_cuda(torch::Tensor cost) {
  auto sizes = cost.sizes();

  TORCH_CHECK(sizes[2] >= sizes[1], "The number of tasks must be greater or equal to the number of workers.");

  auto device = cost.device();
  auto matching_options = torch::TensorOptions()
    .dtype(torch::kLong)
    .device(device.type(), device.index());
  torch::Tensor matching = torch::empty({sizes[0], sizes[1]}, matching_options);

  // If sizes[2] is zero, then sizes[1] is also zero.
  if (sizes[0] * sizes[1] == 0) {
    return matching;
  }

  AT_DISPATCH_FLOATING_TYPES(cost.type(), "solve_cuda_batch", ([&] {
    solve_cuda_batch<scalar_t, long>(
        sizes[0], sizes[1], sizes[2],
	cost.data<scalar_t>(),
	matching.data<long>());
  }));
  return matching;
}
